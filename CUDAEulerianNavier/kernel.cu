#include "hip/hip_runtime.h"

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <vector>

#include "GLFW\glfw3.h"
#pragma comment(lib, "opengl32.lib")
#pragma comment(lib, "GLFW/glfw3dll.lib")

#include <glm\glm.hpp>

#include <cusp\array1d.h>
#include <cusp\coo_matrix.h>
#include <cusp\krylov\bicgstab.h>
#include <cusp\krylov\cg.h>
#include <cusp\monitor.h>
#include <cusp\precond\diagonal.h>

#include <Windows.h>

#include "fluidQ.h"

using namespace std;
using namespace glm;

const float imageWidth = 600,
imageHeight = 600;

float PI = 3.14159;

const int mapW = 128,
mapH = 128;

#define dt (1.f / 30.f)
const float rho = 0.1;

float p[mapW][mapH];

GLFWwindow* window;

enum cellType
{
	WATER, AIR, SOLID
};
cellType type[mapW * mapH];

vector<vec2> parts;

fluidQ* u;
fluidQ* v;

fluidQ u_device, v_device;
vec2* parts_device;
cellType* type_device;

//-----------------------------------------------------------------------------
// device functions
//-----------------------------------------------------------------------------
__device__ int getIdx()
{
	return threadIdx.x + blockDim.x*blockIdx.x;
}

//-----------------------------------------------------------------------------
// global functions
//-----------------------------------------------------------------------------
__global__ void clearCellType(cellType* type)
{
	int idx = getIdx();

	if (idx >= mapW * mapH)
		return;

	if (type[idx] != SOLID)
		type[idx] = AIR;
}
__global__ void updateParticles(fluidQ u, fluidQ v, vec2* parts, int numParts, cellType* type)
{
	int idx = getIdx();

	if (idx >= numParts)
		return;

	float uVel = u.lerp(parts[idx].x, parts[idx].y);
	float vVel = v.lerp(parts[idx].x, parts[idx].y);

	parts[idx].x += uVel * dt;
	parts[idx].y += vVel * dt;

	if (parts[idx].x < 0)
		parts[idx].x = 0;
	if (parts[idx].y < 0)
		parts[idx].y = 0;
	if (parts[idx].x > mapW - 0.01)
		parts[idx].x = mapW - 0.01;
	if (parts[idx].y > mapH - 0.01)
		parts[idx].y = mapH - 0.01;
	
	if (type[(int)parts[idx].y * mapW + (int)parts[idx].x] != SOLID)
		type[(int)parts[idx].y * mapW + (int)parts[idx].x] = WATER;
}

//-----------------------------------------------------------------------------
// host methods
//-----------------------------------------------------------------------------
float nrand()
{
	return (float)rand() / RAND_MAX;
}

void spawnUniformParticles(int x, int y, float n)
{
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			parts.push_back(vec2(x + nrand() / n + i / n, y + nrand() / n + j / n));
		}
	}
}

void setupParticles()
{
	for (int x = 0; x < mapW + 1; ++x)
	{
		for (int y = 0; y < mapH + 1; ++y)
		{
			u->set(x, y, 0);
			v->set(x, y, 0);
		}
	}

	for (int x = 0; x < mapW / 4; ++x)
	{
		for (int y = 0; y < mapH; ++y)
		{
			spawnUniformParticles(x, y, 2);
		}
	}
}
void createWalls()
{
	for (int y = 0; y < mapH; ++y)
	{
		type[y * mapW + 25] = SOLID;
		type[y * mapW + mapW - 10] = SOLID;
	}
	for (int i = 0; i <= 64; ++i)
	{
		type[(64 - i) * mapW + i] = SOLID;
		if (i < 64) 
			type[(64 - i - 1) * mapW + i] = SOLID;
	}
}

void applyExternal()
{
	for (int y = 0; y < mapH + 1; ++y)
		for (int x = 0; x < mapW; ++x)
			v->at(x, y) -= 9 * dt;
}
void clearCellType()
{
	int blocksize = 1024;
	int numBlocks = mapW * mapH / blocksize + 1;

	hipMemcpy(type_device, &type, mapW * mapH * sizeof(cellType), hipMemcpyHostToDevice);

	clearCellType <<< numBlocks, blocksize >>> (type_device);
	hipDeviceSynchronize();
}
void updateParticles()
{
	int blocksize = 1024;
	int numBlocks = parts.size() / blocksize + 1;

	hipMemcpy(u_device.cur, u->cur, u->w * u->h * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(v_device.cur, v->cur, v->w * v->h * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(parts_device, &parts[0], parts.size() * sizeof(vec2), hipMemcpyHostToDevice);

	updateParticles <<< numBlocks, blocksize >>> (u_device, v_device, parts_device, parts.size(), type_device);
	hipDeviceSynchronize();

	hipMemcpy(&parts[0], parts_device, parts.size() * sizeof(vec2), hipMemcpyDeviceToHost);
	hipMemcpy(&type, type_device, mapW * mapH * sizeof(cellType), hipMemcpyDeviceToHost);
}
int layerField[mapW][mapH];
void extrapolate()
{
	for (int y = 0; y < mapH; ++y)
	{
		for (int x = 0; x < mapW; ++x)
		{
			if (type[y * mapW + x] == WATER)
				layerField[x][y] = 0;
			else
				layerField[x][y] = -1;
		}
	}

	for (int i = 1; i < 6; ++i)
	{
		for (int y = 0; y < mapH; ++y)
		{
			for (int x = 0; x < mapW; ++x)
			{
				if (layerField[x][y] != -1)
					continue;

				bool l, t, r, b;
				l = t = r = b = false;
				float uAvg = 0; int uN = 0;
				float vAvg = 0; int vN = 0;
				if (x > 0)
				{
					if (layerField[x - 1][y] == i - 1)
					{
						l = true;
						uAvg += u->at(x, y);
						++uN;
					}
				}
				if (y > 0)
				{
					if (layerField[x][y - 1] == i - 1)
					{
						t = true;
						vAvg += v->at(x, y);
						++vN;
					}
				}
				if (x < mapW - 1)
				{
					if (layerField[x + 1][y] == i - 1)
					{
						r = true;
						uAvg += u->at(x + 1, y);
						++uN;
					}
				}
				if (y < mapH - 1)
				{
					if (layerField[x][y + 1] == i - 1)
					{
						b = true;
						vAvg += v->at(x, y + 1);
						++vN;
					}
				}

				if (!(l || t || r || b))
					continue;

				uAvg = uAvg / max(1, uN);
				vAvg = vAvg / max(1, vN);

				if (x > 0)
				{
					if (type[y * mapW + x - 1] != WATER)
					{
						u->at(x, y) = uAvg;
					}
				}
				if (y > 0)
				{
					if (type[(y - 1) * mapW + x] != WATER)
					{
						v->at(x, y) = vAvg;
					}
				}
				if (x < mapW - 1)
				{
					if (type[y * mapW + x + 1] != WATER)
					{
						u->at(x + 1, y) = uAvg;
					}
				}
				if (y < mapH - 1)
				{
					if (type[(y + 1) * mapW + x] != WATER)
					{
						v->at(x, y + 1) = vAvg;
					}
				}

				layerField[x][y] = i;
			}
		}
	}
}

vector<int> reposistion;
int numParts[mapW][mapH];
void computeReposition()
{
	reposistion.clear();
	for (int y = 0; y < mapH; ++y)
	{
		for (int x = 0; x < mapW; ++x)
		{
			numParts[x][y] = 0;
		}
	}

	for (int i = 0; i < parts.size(); ++i)
	{
		vec2 p = parts[i];
		int x = p.x;
		int y = p.y;

		if (type[y * mapW + x] != SOLID)
			numParts[x][y] += 1;

		if (type[y * mapW + x] == SOLID)
			reposistion.push_back(i);
		if (numParts[x][y] > 8)
			reposistion.push_back(i);
	}
}
void repositionParticles()
{
	for (int y = 0; y < mapH; ++y)
	{
		for (int x = 0; x < mapW; ++x)
		{
			if (type[y * mapW + x] == SOLID)
				continue;

			boolean isLonely = false;
			isLonely &= (type[y * mapW + x] == AIR);
			if (x > 0)
				isLonely &= (type[y * mapW + x - 1] == WATER);
			if (y > 0)
				isLonely &= (type[(y - 1) * mapW + x] == WATER);
			if (x < mapW - 1)
				isLonely &= (type[y * mapW + x + 1] == WATER);
			if (y < mapH - 1)
				isLonely &= (type[(y + 1) * mapW + x] == WATER);

			if (numParts[x][y] == 0 && !isLonely)
				continue;

			while (reposistion.size() != 0 && numParts[x][y] < 4)
			{
				int i = reposistion.back();
				reposistion.pop_back();

				parts[i].x = x + nrand();
				parts[i].y = y + nrand();

				numParts[x][y] += 1;

				type[y * mapW + x] = WATER;
			}
		}
	}
}

struct cuspTriple
{
	int row, col;
	float amount;
};
//int countBuffer[mapW][mapH];
cusp::array1d<float, cusp::host_memory> pressure(mapW * mapH);
void project()
{
	/*{
		int counter = 0;
		for (int y = 0; y < mapH; ++y)
		{
			for (int x = 0; x < mapW; ++x)
			{
				if (type[y * mapW + x] == WATER)
				{
					countBuffer[x][y] = counter;
					++counter;
				}
				else
					countBuffer[x][y] = -1;
			}
		}
	}*/
	
	cusp::array1d<float, cusp::host_memory> b(mapW * mapH);
	{
		float scale = rho / dt;
		for (int y = 0; y < mapH; ++y)
		{
			for (int x = 0; x < mapW; ++x)
			{
				/*int index = countBuffer[x][y];
				if (index == -1)
					continue;*/
				int index = y * mapW + x;
				
				/*if (type[y * mapW + x] != WATER)
				{
					b[index] = 0;
					continue;
				}*/

				b[index] = scale * (u->at(x + 1, y) - u->at(x, y) +
					v->at(x, y + 1) - v->at(x, y));
			}
		}
	}

	vector<cuspTriple> data;
	{
		for (int y = 0; y < mapH; ++y)
		{
			for (int x = 0; x < mapW; ++x)
			{
				float scale = 1;
				int n = 0;

				/*if (type[y * mapW + x] != WATER)
				{
					continue;
				}*/

				if (x > 0) 
				{
					if (type[y * mapW + x - 1] != SOLID)
					{
						if (type[y * mapW + x - 1] == WATER)
						{
							cuspTriple t;
							t.row = y * mapW + x;
							t.col = y * mapW + x - 1;
							t.amount = 1;
							data.push_back(t);
						}
						++n;
					}
				}
				if (y > 0) {
					if (type[(y - 1) * mapW + x] != SOLID)
					{
						if (type[(y - 1) * mapW + x] == WATER)
						{
							cuspTriple t;
							t.row = y * mapW + x;
							t.col = (y - 1) * mapW + x;
							t.amount = 1;
							data.push_back(t);
						}
						++n;
					}
				}
				if (x < mapW - 1) {
					if (type[y * mapW + x + 1] != SOLID)
					{
						if (type[y * mapW + x + 1] == WATER)
						{
							cuspTriple t;
							t.row = y * mapW + x;
							t.col = y * mapW + x + 1;
							t.amount = 1;
							data.push_back(t);
						}
						++n;
					}
				}
				if (y < mapH - 1) {
					if (type[(y + 1) * mapW + x] != SOLID)
					{
						if (type[(y + 1) * mapW + x] == WATER)
						{
							cuspTriple t;
							t.row = y * mapW + x;
							t.col = (y + 1) * mapW + x;
							t.amount = 1;
							data.push_back(t);
						}
						++n;
					}
				}

				cuspTriple t;
				t.row = y * mapW + x;
				t.col = y * mapW + x;
				t.amount = -n;
				data.push_back(t);
			}
		}

	}
	cusp::coo_matrix<int, float, cusp::host_memory> A(mapW * mapH, mapW * mapH, data.size());
	{
		for (int i = 0; i < data.size(); ++i)
		{
			A.row_indices[i] = data[i].row;
			A.column_indices[i] = data[i].col;
			A.values[i] = data[i].amount;
		}
	}

	cusp::default_monitor<float> monitor(b, 600, 0.01, 0);
	cusp::precond::diagonal<float, cusp::host_memory> M(A);

	cusp::krylov::cg(A, pressure, b, monitor, M);
}
void applyPressure()
{
	float scale = dt / (rho);

	for (int y = 0; y < mapH; y++)
	{
		for (int x = 0; x < mapW; x++)
		{
			if (type[y * mapW + x] != WATER)
				continue;

			float p = pressure[y * mapW + x];

			u->at(x, y) -= scale * p;
			u->at(x + 1, y) += scale * p;
			v->at(x, y) -= scale * p;
			v->at(x, y + 1) += scale * p;
		}
	}
}

void enforceBoundary()
{
	for (int y = 0; y < mapH; ++y)
	{
		for (int x = 0; x < mapW; ++x)
		{
			if (type[y * mapW + x] != SOLID)
				continue;

			u->at(x, y) = 0;
			u->at(x + 1, y) = 0;

			v->at(x, y) = 0;
			v->at(x, y + 1) = 0;
		}
	}

	for (int x = 0; x < mapW + 1; ++x)
	{
		v->set(x, 0, 0);
		v->set(x, v->h - 1, 0);
	}
	for (int y = 0; y < mapH + 1; ++y)
	{
		u->set(0, y, 0);
		u->set(u->w - 1, y, 0);
	}
}

void update()
{
	/*static int iter = 0;
	if (iter > 800)
	{
		return;
	}
	++iter;
	printf("%d\n", iter);*/
	
	applyExternal();

	clearCellType();
	updateParticles();
	extrapolate();

	computeReposition();
	repositionParticles();

	project();
	applyPressure();

	enforceBoundary();

	u->advect(dt, u, v);
	v->advect(dt, u, v);

	u->flip();
	v->flip();

	//printf("%d\n", (int)parts.size());

	// diagnostics
	{
		double rx, ry;
		glfwGetCursorPos(window, &rx, &ry);
		rx /= imageWidth / mapW; ry /= imageHeight / mapH;
		ry = mapH - ry;

		//printf("%i, %i \n", (int)rx, (int)ry);

		if (0 <= rx && rx < mapW &&
			0 <= ry && ry < mapH)
		{
			printf("-----------------------------\n");
			printf("V: %f, %f\n", u->lerp(rx, ry), v->lerp(rx, ry));
			printf("P: %f\n", p[(int)rx][(int)ry]);
			printf("-----------------------------\n\n");
		}

		if (0 <= rx && rx < mapW &&
			0 <= ry && ry < mapH)
		{
			if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT))
			{
				//ink->set(rx, ry, ink->at(rx, ry) + 1);
				//vel[(int)rx][(int)ry] += p;
				//printf("clicalsdhfalskdfuahsdkxlfajh\n");

				for (int y = 0; y < mapH; ++y)
				{
					for (int x = 0; x < mapW; ++x)
					{
						float ix = (int)rx + 0.5f;
						float iy = (int)ry + 0.5f;

						vec2 r = vec2(ix, iy) - vec2(x, y);
						float t = 500 / dot(r, r);
						r = normalize(r);

						u->at(x, y) += r.x * t;
						v->at(x, y) += r.y * t;
					}
				}

			}
		}
	}
}

void draw()
{
	glViewport(0, 0, imageWidth, imageHeight);
	glClear(GL_COLOR_BUFFER_BIT);

	glLoadIdentity();
	glTranslatef(-1, -1, 0);
	glScalef(2, 2, 1);
	glScalef(1.f / mapW, 1.f / mapH, 1);

	/*glBegin(GL_QUADS);
	{
		for (int x = 0; x < mapW; ++x)
		{
			for (int y = 0; y < mapH; ++y)
			{
				float f = numParts[x][y] / 4.f;
				glColor3f(0, 0, f);
				//glColor3f(0, 0, 0);
				if (type[y * mapW + x] == SOLID)
					glColor3f(0, 1, 0);

				glVertex2f(x, y);
				glVertex2f(x + 1, y);
				glVertex2f(x + 1, y + 1);
				glVertex2f(x, y + 1);
			}
		}
	}
	glEnd();*/

	glColor3f(1, 0, 0);
	glBegin(GL_POINTS);
	{
		for (vec2 p : parts)
		{
			glVertex2f(p.x, p.y); 
		}
	}
	glEnd();
}

int main()
{
	srand(time(0));

	if (!glfwInit())
	{
		printf("couldn't initialize GLFW");
		return 0;
	}

	// no window hints. don't really care

	window = glfwCreateWindow(imageWidth, imageHeight, "lulz", NULL, NULL);
	if (!window)
	{
		printf("failed to open glfw window");
		return 0;
	}

	glfwMakeContextCurrent(window);
	glfwSwapInterval(1);

	glEnable(GL_BLEND);
	glBlendFunc(GL_SRC_ALPHA, GL_ONE);

	u = new fluidQ();
	u->create(mapW + 1, mapH + 1, 0.0, 0.5, 1);
	v = new fluidQ();
	v->create(mapW + 1, mapH + 1, 0.5, 0.0, 1);
	setupParticles();

	memset(p, 0, mapW * mapH * sizeof(float));
	for (int y = 0; y < mapH; ++y)
	{
		for (int x = 0; x < mapW; ++x)
		{
			type[y * mapW + x] = AIR;
		}
	}
	//createWalls();

	int maxParts = 70000;
	u_device.createCUDA(u->w, u->h, u->ox, u->oy, u->delta_x);
	v_device.createCUDA(v->w, v->h, v->ox, v->oy, v->delta_x);
	hipMalloc((void**)&parts_device, maxParts * sizeof(vec2));
	hipMalloc((void**)&type_device, mapW * mapH * sizeof(cellType));

	// main loop
	auto currentTime = chrono::high_resolution_clock::now();
	float accumulator = 0;
	int iter = 0;
	while (!glfwWindowShouldClose(window))
	{
		auto newTime = chrono::high_resolution_clock::now();
		float frameTime = chrono::duration_cast<chrono::milliseconds>(newTime - currentTime).count();

		if (frameTime >= dt)
		{
			update();

			++iter;
			//printf("%d\n", iter);

			currentTime = newTime;
		}

		draw();

		//Sleep(500);

		//printf("%d\n", window);

		glfwSwapBuffers(window);
		glfwPollEvents();
	}
	printf("%d", window);
	glfwDestroyWindow(window);
	glfwTerminate();

	delete u;
	delete v;

	return 0;
}